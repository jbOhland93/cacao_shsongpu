#include "SGE_Evaluator.hpp"
#include "milkDebugTools.h"

#include <hip/hip_runtime.h>
#include <math.h>

#include "SGE_GridLayout.hpp"
#include "SGE_Kernel.hpp"

#include <chrono>

using namespace std::chrono;

SGE_Evaluator::SGE_Evaluator(
        IMAGE* in,          // Raw camera stream
        IMAGE* dark,        // Stream holding a dark for subtraction
        int deviceID)       // ID of the GPU
    : mp_im(in), m_deviceID(deviceID)
{
    hipError_t err;
    err = hipSetDevice(m_deviceID);
    printCE(err);
    err = hipSetDeviceFlags(hipDeviceMapHost);
    printCE(err);

    copyDarkToGPU(dark);

    // Create some tile coordinates, which ultimately live on the GPU forever
    uint16_t tileGridSize = 20;
    int bufSize = tileGridSize*tileGridSize*sizeof(uint16_t);
    hipHostAlloc((void**)&m_hp_ROIx, bufSize, hipHostMallocMapped);
    hipHostAlloc((void**)&m_hp_ROIy, bufSize, hipHostMallocMapped);
    for (uint16_t x = 0; x < tileGridSize; x++)
        for (uint16_t y = 0; y < tileGridSize; y++)
        {
            int xPre = x * 16 + rand()%3-1 + 20; // randomize;
            m_hp_ROIx[x+y*tileGridSize] = xPre < 0 ? 0 : xPre;
            int yPre = y * 16 + rand()%3-1 + 20; // randomize;
            m_hp_ROIy[x+y*tileGridSize] = yPre < 0 ? 0 : yPre;
        }
    hipMalloc((void**)&m_dp_ROIx, bufSize);
    hipMalloc((void**)&m_dp_ROIy, bufSize);
    hipMemcpy(m_dp_ROIx, m_hp_ROIx, bufSize, hipMemcpyHostToDevice);
    hipMemcpy(m_dp_ROIy, m_hp_ROIy, bufSize, hipMemcpyHostToDevice);

    // Create a block kernel, which ultimately lives on the GPU
    int kernelSize = 3;
    float* h_kernel;
    hipHostAlloc((void**)&h_kernel, kernelSize*kernelSize*sizeof(float), hipHostMallocMapped);
    for (uint16_t x = 0; x < kernelSize; x++)
        for (uint16_t y = 0; y < kernelSize; y++)
        {
            if (x==1 && y==1)
                h_kernel[x+y*kernelSize] = 0.111;
            else
                h_kernel[x+y*kernelSize] = 0.111;
        }
    hipMalloc((void**)&m_dp_kernel, kernelSize*kernelSize*sizeof(float));
    hipMemcpy(m_dp_kernel, h_kernel, kernelSize*kernelSize*sizeof(float), hipMemcpyHostToDevice);
    hipHostFree(h_kernel);

    mpGridLayout = std::make_shared<SGE_GridLayout>(m_deviceID, tileGridSize*tileGridSize, kernelSize, 4);
}

errno_t SGE_Evaluator::evaluateDo()
{
    // ==== Prepare timing utilities
    hipEvent_t start, stop;
    float time;
    hipError_t err = hipEventCreate(&start);
    printCE(err);
    err = hipEventCreate(&stop);
    printCE(err);
    // ====



    printf("\nStart evaluateDo\n\n");
    // Prepare image buffer
    // Ultimately, the image source should already be a mapped pointer
    uint16_t* readBufBrightNonMapped;
    ImageStreamIO_readLastWroteBuffer(mp_im, (void**)&readBufBrightNonMapped);
    uint16_t* h_readBufBright;
    int bufSize = mp_im->md->size[0]*mp_im->md->size[1]*sizeof(uint16_t);
    err = hipHostAlloc(&h_readBufBright, bufSize, hipHostMallocMapped);
    printCE(err);
    err = hipMemcpy((void**)h_readBufBright, readBufBrightNonMapped, bufSize, hipMemcpyHostToHost);
    printCE(err);

    // Prepare dark buffer - the image already is on the GPU.
    float* d_readBufDark;
    ImageStreamIO_readLastWroteBuffer(&m_imDarkGPU, (void**)&d_readBufDark);

    // Prepare result buffer
    float* h_intensity;
    float* d_intensity;
    err = hipHostAlloc(&h_intensity, mpGridLayout->mNumSubapertures*sizeof(float), hipHostMallocMapped);
    printCE(err);
    err = hipMalloc(&d_intensity, mpGridLayout->mNumSubapertures*sizeof(float));
    printCE(err);
    
    float* h_convImage;
    IMAGE convImage;
    ImageStreamIO_openIm(&convImage, "ximeaCam_AVG2");
    ImageStreamIO_writeBuffer(&convImage, (void**)&h_convImage);

    float* h_darkPx;
    hipHostAlloc(&h_darkPx, m_imDarkGPU.md->imdatamemsize, hipHostMallocMapped);
    hipMemcpy(h_darkPx, d_readBufDark, m_imDarkGPU.md->imdatamemsize, hipMemcpyDeviceToHost);
    for (int i = 0; i < mp_im->md->size[0]*mp_im->md->size[1]; i++)
    {
        h_convImage[i] = (float) h_readBufBright[i] - h_darkPx[i];
    }
    hipHostFree(h_darkPx);
    ImageStreamIO_UpdateIm(&convImage);
    float* d_convImage;
    hipMalloc(&d_convImage, convImage.md->imdatamemsize);
    hipMemcpy(d_convImage, h_convImage, convImage.md->imdatamemsize, hipMemcpyHostToDevice);

    err = hipEventRecord( start, 0 );
    printCE(err);

    // Do processing
/*  evaluateSpot<<<
        mpGridLayout->mNumSubapertures,
        mpGridLayout->mBlockSize,
        mpGridLayout->mShmSize>>>(
            h_readBufBright,
            d_readBufDark,
            m_dp_kernel,
            mp_im->md->size[0],
            m_dp_ROIx,
            m_dp_ROIy,
            mpGridLayout->getDeviceCopy(),
            d_intensity,
            d_convImage);
*/
// ============================================
    hipStream_t streams[mpGridLayout->mNumSubapertures];
    for (int i = 0; i < mpGridLayout->mNumSubapertures; ++i)
        hipStreamCreate(&streams[i]);
    
    int blockSize = mpGridLayout->mKernelSize;
    blockSize *= blockSize;
    blockSize *= mpGridLayout->mNumCorrelPosPerAp;
    blockSize /= 32;
    blockSize++;
    blockSize *= 32;

    int numPxInWindow = mpGridLayout->mNumWindowPx;
    int numPxInKernel = mpGridLayout->mNumKernelPx;
    int numCorrelPos = mpGridLayout->mNumCorrelPosPerAp;
    int numCorrelOps = numCorrelPos * numPxInKernel;
    int shmSize = numPxInWindow + numPxInKernel + 2*numCorrelPos + numCorrelOps;
    shmSize *= sizeof(float);

    int convCoordsX[numCorrelPos];
    int convCoordsY[numCorrelPos];
    mpGridLayout->gnrtCorrelOffsetsFrmRootsHost(convCoordsX, convCoordsY);
    int* d_convCoordsX;
    int* d_convCoordsY;
    hipMalloc(&d_convCoordsX, numCorrelPos*sizeof(int));
    hipMalloc(&d_convCoordsY, numCorrelPos*sizeof(int));
    hipMemcpy(d_convCoordsX, convCoordsX, numCorrelPos*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_convCoordsY, convCoordsY, numCorrelPos*sizeof(int), hipMemcpyHostToDevice);

    float* d_debug;
    int debugBufSize = 128;
    hipMalloc(&d_debug, debugBufSize*sizeof(float));

    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    /*One launch per block
    for (int i = 0; i < mpGridLayout->mNumSubapertures; ++i)
        evaluateSingleSpot<<<1, blockSize, shmSize, streams[i]>>>(
            h_readBufBright,                //uint16_t* h_imageData,
            d_readBufDark,                  //float* d_darkData,
            d_convImage,                    //float* d_outputImage,
            mp_im->md->size[0],             //int imW,
            m_hp_ROIx[i],                   //int windowRootX,
            m_hp_ROIy[i],                   //int windowRootY,
            mpGridLayout->mWindowSize,      //int windowSize,
            m_dp_kernel,                    //float* d_kernel,
            mpGridLayout->mKernelSize,      //int kernelSize,
            d_convCoordsX,                  //int* d_convCoordsX,
            d_convCoordsY,                  //int* d_convCoordsY,
            mpGridLayout->mNumCorrelPosPerAp//int numConvCoords)
            );
    */
    evaluateSpots<<<mpGridLayout->mNumSubapertures, blockSize, shmSize>>>(
            h_readBufBright,                    //uint16_t* h_imageData,
            d_readBufDark,                      //float* d_darkData,
            d_convImage,                        //float* d_outputImage,
            mp_im->md->size[0],                 //int imW,
            m_dp_ROIx,                          //int windowRootX,
            m_dp_ROIy,                          //int windowRootY,
            mpGridLayout->mWindowSize,          //int windowSize,
            m_dp_kernel,                        //float* d_kernel,
            mpGridLayout->mKernelSize,          //int kernelSize,
            d_convCoordsX,                      //int* d_convCoordsX,
            d_convCoordsY,                      //int* d_convCoordsY,
            mpGridLayout->mNumCorrelPosPerAp,   //int numConvCoords,
            mpGridLayout->mCorrelMargin,        //int correlMargin
            d_debug                             //float* debug
            );
    
    hipDeviceSynchronize();
    high_resolution_clock::time_point t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    printf("Time for kernel call: %.3f µs\n", time_span.count()*1e6);
    
    float* h_debug;
    hipHostAlloc(&h_debug, debugBufSize*sizeof(float), hipHostMallocMapped);
    hipMemcpy(h_debug, d_debug, debugBufSize*sizeof(float), hipMemcpyDeviceToHost);
    printf("Debug 0: %.3f\n", h_debug[0]);
    printf("Debug 1: %.3f\n", h_debug[1]);
    printf("Debug 2: %.3f\n", h_debug[2]);
    printf("Debug 3: %.3f\n", h_debug[3]);
    printf("Debug 4: %.3f\n", h_debug[4]);
    printf("Debug 5: %.3f\n", h_debug[5]);
    printf("Debug 6: %.3f\n", h_debug[6]);
    printf("Debug 7: %.3f\n", h_debug[7]);
    printf("Debug 8: %.3f\n", h_debug[8]);
    printf("Debug 9: %.3f\n", h_debug[9]);
    
    
// ============================================

    err = hipEventRecord( stop, 0 );
    printCE(err);

    // Get timing
    err = hipEventSynchronize( stop );
    printCE(err);
    err = hipEventElapsedTime( &time, start, stop );
    printCE(err);
    //printf("Time for kernel call: %.3f µs\n", time*1000);

    // Print error of kernel launch
    printf("Assessing errors of kernel launch ... ");
    //err = hipDeviceSynchronize();
    //printCE(err);
    err = hipGetLastError();
    printCE(err);
    printf("Assessing done!\n");

    // Copy result
    /*err = hipMemcpy(h_intensity, d_intensity, mpGridLayout->mNumSubapertures*sizeof(float), hipMemcpyDeviceToHost);
    printCE(err);
    printf("CorrelOPs pT = %.3f\n", h_intensity[5]);
    printf("CorrelPos pA = %.3f\n\n", h_intensity[6]);
    printf("Thread Index = %.3f\n", h_intensity[0]);
    printf("Apertr Index = %.3f\n", h_intensity[1]);
    printf("Coordn Index = %.3f\n", h_intensity[2]);
    printf("KernlY Index = %.3f\n", h_intensity[4]);
    printf("KernlX Index = %.3f\n\n", h_intensity[3]);*/

    // Copy convoluted image and post it to ISIO
    err = hipMemcpy(h_convImage, d_convImage, convImage.md->imdatamemsize, hipMemcpyDeviceToHost);
    printCE(err);
    ImageStreamIO_UpdateIm(&convImage);

    // Clean up
    hipHostFree(h_readBufBright);
    hipHostFree(h_intensity);
    hipFree(d_intensity);
    hipFree(d_convImage);
    hipFree(d_convCoordsX);
    hipFree(d_convCoordsY);
    hipFree(d_debug);
    hipHostFree(h_debug);
    err = hipEventDestroy( start );
    printCE(err);
    err = hipEventDestroy( stop );
    printCE(err);

    return RETURN_SUCCESS;
}

SGE_Evaluator::~SGE_Evaluator()
{
    // Delete the darkframe from the GPU
    ImageStreamIO_destroyIm(&m_imDarkGPU);
    // Free array
    hipHostFree(m_hp_ROIx);
    hipHostFree(m_hp_ROIy);
    hipFree(m_dp_ROIx);
    hipFree(m_dp_ROIy);
    hipFree(m_dp_kernel);
}

void SGE_Evaluator::copyDarkToGPU(IMAGE* dark)
{
    std::string imName(dark->name);
    imName.append("_GPU");

    ImageStreamIO_createIm_gpu(
        &m_imDarkGPU,
        imName.c_str(),
        dark->md->naxis,
        dark->md->size,
        dark->md->datatype,
        m_deviceID,         // -1: CPU RAM, 0+ : GPU
        1,                  // shared?
        0,                  // # of semaphores
        0,                  // # of keywords
        dark->md->imagetype,
        0 // circular buffer size (if shared), 0 if not used
    );

    float* readBuf;
    ImageStreamIO_readLastWroteBuffer(dark, (void**)&readBuf);
    float* writeBuf;
    ImageStreamIO_writeBuffer(&m_imDarkGPU, (void**)&writeBuf);
    
    hipError_t err;
    err = hipMemcpy(writeBuf, readBuf, dark->md->imdatamemsize,
        hipMemcpyHostToDevice);
    printCE(err);
    ImageStreamIO_UpdateIm(&m_imDarkGPU);

    
}
