#include "ImageHandler2DBase.hpp"

#include <hip/hip_runtime.h>
#include "CudaUtil.hpp"

ImageHandler2DBase::~ImageHandler2DBase()
{   
    // Destroy the image only if persistent is not enabled.
    if (!mPersistent)
        ImageStreamIO_destroyIm(mp_image);

    // Clean up device memory in any case
    if (mp_d_imData != nullptr)
        hipFree(mp_d_imData);
    
    delete mp_image;
}

hipError_t ImageHandler2DBase::mapImForGPUaccess()
{
    return hipHostRegister(
            ImageStreamIO_get_image_d_ptr(mp_image),
            mp_image->md->imdatamemsize,
            hipHostRegisterMapped);
}

void ImageHandler2DBase::setSlice(uint32_t sliceIndex)
{
    if (sliceIndex >= mDepth)
        throw std::runtime_error("SGR_ImageHandler::setSlice: out of range.");
    else
        m_currentSlice = sliceIndex;
}

void ImageHandler2DBase::setROI(Rectangle<uint32_t> roi)
{
    if (roi.x()+roi.w() >= mWidth || roi.y()+roi.h() >= mHeight)
        throw std::runtime_error("SGR_ImageHandler::setROI: out of range.");
    else
        mROI = roi;
}

void ImageHandler2DBase::setROI(uint32_t x, uint32_t y, uint32_t w, uint32_t h)
{
    setROI(Rectangle<uint32_t>(x,y,w,h));
}

void ImageHandler2DBase::unsetROI()
{
    mROI = Rectangle<uint32_t>(0,0, mWidth, mHeight);
}

ImageHandler2DBase::ImageHandler2DBase(
        uint32_t width,
        uint32_t height,
        uint32_t depth)
        :
        mWidth(width),
        mHeight(height),
        mDepth(depth),
        mNumPx(width*height*depth),
        mROI(0,0,width,height)
{
    mp_image = new IMAGE();
}

void ImageHandler2DBase::updateImMetadata()
{
    mp_h_imData = ImageStreamIO_get_image_d_ptr(mp_image);
    m_dataSize = mp_image->md->imdatamemsize;
}

void* ImageHandler2DBase::getDeviceCopy()
{
    if (m_gpuCopySize != mp_image->md->imdatamemsize)
        updateDeviceCopy();
    return mp_d_imData;
}

void ImageHandler2DBase::updateDeviceCopy()
{
    hipError_t err;
    if (m_gpuCopySize != mp_image->md->imdatamemsize && mp_d_imData != nullptr) 
    {
            err = hipFree(mp_d_imData);
            printCE(err);
            mp_d_imData = nullptr;
    }
    if (mp_d_imData == nullptr)
    {
        m_gpuCopySize = mp_image->md->imdatamemsize;
        err = hipMalloc((void**)&mp_d_imData, m_gpuCopySize);
        printCE(err);
    }
    // Only perform the copy if the host image has been updated
    if (mp_image->md->cnt0 != mCnt0deviceCopy)
    {
        err = hipMemcpy(
            mp_d_imData,
            mp_h_imData,
            m_gpuCopySize,
            hipMemcpyHostToDevice);
        printCE(err);
        mCnt0deviceCopy = mp_image->md->cnt0;
    }
}

void ImageHandler2DBase::updateFromDevice()
{
    if (mp_d_imData == nullptr)
        throw std::runtime_error("ImageHandler2DBase::updateFromDevice: No device copy used.\n");
    if (m_gpuCopySize != mp_image->md->imdatamemsize)
        throw std::runtime_error("ImageHandler2DBase::updateFromDevice: Array size mismatch.\n");
    
    void* dst;
    ImageStreamIO_readLastWroteBuffer(mp_image, &dst);
    hipError_t err;
    err = hipMemcpy(
        dst,
        mp_d_imData,
        m_gpuCopySize,
        hipMemcpyDeviceToHost);
    printCE(err);
    ImageStreamIO_UpdateIm(mp_image);
}

uint32_t ImageHandler2DBase::fromROIxToImX(uint32_t x)
{
    if (x >= mROI.w()) // x is uint3_t, thus always > 0
        throw std::runtime_error("ImageHandler2DBase::toROIx: x is out of range.");
    else
        return x + mROI.x();
}

uint32_t ImageHandler2DBase::fromROIyToImY(uint32_t y)
{
    if (y >= mROI.h()) // y is uint3_t, thus always > 0
        throw std::runtime_error("ImageHandler2DBase::toROIy: y is out of range.");
    else
        return y + mROI.y();
}

std::vector<uint32_t> ImageHandler2DBase::getSizeVector()
{
    return std::vector<uint32_t>({mWidth, mHeight, mDepth});
}

int ImageHandler2DBase::getKWindex(std::string name)
{
    for (int i = 0; i < mp_image->md->NBkw; i++)
    {
        std::string kwName = mp_image->kw[i].name;
        while(kwName.length() > name.length())
            kwName.pop_back();
        if (name == kwName)
            return i;
    }
    return -1;
}